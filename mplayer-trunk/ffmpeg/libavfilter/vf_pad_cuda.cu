
#include <hip/hip_runtime.h>
/*
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

template <typename T>
__device__ void pad_impl(T* dst, int dst_pitch, int dst_w, int dst_h,
                         const T* src, int src_pitch, int src_w, int src_h,
                         int roi_x, int roi_y, T fill_val)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst_w || y >= dst_h) {
        return;
    }

    if (x >= roi_x && x < (roi_x + src_w) && y >= roi_y && y < (roi_y + src_h)) {
        const int src_x = x - roi_x;
        const int src_y = y - roi_y;
        dst[y * dst_pitch + x] = src[src_y * src_pitch + src_x];
    } else {
        dst[y * dst_pitch + x] = fill_val;
    }
}


extern "C" {

__global__ void pad_uchar(unsigned char* dst, int dst_pitch, int dst_w, int dst_h,
                          const unsigned char* src, int src_pitch, int src_w, int src_h,
                          int roi_x, int roi_y, unsigned char fill_val)
{
    pad_impl<unsigned char>(dst, dst_pitch, dst_w, dst_h,
                            src, src_pitch, src_w, src_h,
                            roi_x, roi_y, fill_val);
}

__global__ void pad_uchar2(uchar2* dst, int dst_pitch, int dst_w, int dst_h,
                           const uchar2* src, int src_pitch, int src_w, int src_h,
                           int roi_x, int roi_y, uchar2 fill_val)
{
    pad_impl<uchar2>(dst, dst_pitch, dst_w, dst_h,
                     src, src_pitch, src_w, src_h,
                     roi_x, roi_y, fill_val);
}

}
